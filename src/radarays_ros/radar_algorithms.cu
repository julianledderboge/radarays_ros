#include "hip/hip_runtime.h"
#include "radarays_ros/radar_algorithms.cuh"
#include <iostream>

namespace radarays_ros
{


/**
 * @brief computes the total energy of back reflection
 * 
 * @param incidence_angle radian
 * @param energy 
 * @param diffuse 
 * @param specular 
 * @return float 
 */

__device__ __forceinline__
float back_reflection_shader(
    float incidence_angle, 
    float energy,
    float diffuse, // A
    float specular_fac, // B
    float specular_exp) // C
{
    // Diffuse: Lambertian model - NdotL
    // NdotL = v*w. for our case v*w = ||v|| * ||w|| * cos(incidence_angle) = cos(incidence_angle)
    // incidence_angle -> 0 - pi/2
    // I_diffuse 1 -> 0
    float IdotR = cos(incidence_angle);

    float I_diffuse = 1.0;
    float I_specular = pow(IdotR, specular_exp);

    // polynom
    float I_total = diffuse * I_diffuse + specular_fac * I_specular;

    return I_total * energy;
}

__device__ __forceinline__
double get_incidence_angle(
    const rm::Vector& surface_normal,
    const DirectedWave& incidence)
{
    return acos((-incidence.ray.dir).dot(surface_normal));
}

__device__ __forceinline__
double get_incidence_angle(
    const rm::Vector& surface_normal,
    const rm::Vector& incidence_dir)
{
    return acos((-incidence_dir).dot(surface_normal));
}


__global__ 
void propagate_waves_kernel(
    const RadarMaterial* materials,
    const int* object_materials,
    int material_id_air,

    const DirectedWave* waves,
    unsigned int n_waves,
    const uint8_t* res_hits,
    const float* res_ranges, 
    const rm::Vector3* res_normals, 
    const unsigned int* res_object_ids,
    
    Signal* signals,
    DirectedWave* waves_new,
    uint8_t* waves_new_mask)
{
    const float wave_energy_threshold = 0.001;
    const float skip_dist = 0.001;

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int refractionid = tid * 2 + 0;
    unsigned int reflectionid = tid * 2 + 1;

    const uint8_t res_hit = res_hits[tid];
    const float res_range = res_ranges[tid];
    rm::Vector3 surface_normal = res_normals[tid];
    const unsigned int obj_id = res_object_ids[tid];
    

    if(obj_id > 10000)
    {
        // finish and mark
        waves_new_mask[refractionid] = 0;
        waves_new_mask[reflectionid] = 0;
        return;
    }

    if(res_hit == 0)
    {
        // finish and mark
        waves_new_mask[refractionid] = 0;
        waves_new_mask[reflectionid] = 0;
        return;
    }

    // 1. move wave to incident
    DirectedWave incidence = waves[tid];
    {
        incidence.ray.orig = incidence.ray.orig + incidence.ray.dir * res_range;
        incidence.time += res_range / incidence.velocity;
    }

    // 2. split to reflection and refraction
    DirectedWave reflection = incidence;
    DirectedWave refraction = incidence;

    // if wave was in air, switch to new material
    // else if wave was in material, switch to air (is this right ?)
    if(incidence.material_id == material_id_air)
    {
        refraction.material_id = object_materials[obj_id];
    } else {
        refraction.material_id = material_id_air;
    }

    float v_refraction = 1.0;

    if(incidence.material_id != refraction.material_id)
    {
        v_refraction = materials[refraction.material_id].velocity;
    } else {
        v_refraction = incidence.velocity;
    }


    // 3. fresnel
    {
        const double v1 = incidence.velocity;
        const double v2 = v_refraction;

        const double n1 = v2;
        const double n2 = v1;

        // is this correct?
        double incidence_angle = acos((-incidence.ray.dir).dot(surface_normal));

        // reflection
        reflection.ray.orig = incidence.ray.orig;
        reflection.ray.dir = incidence.ray.dir + surface_normal * 2.0 * (-surface_normal).dot(incidence.ray.dir);

        // refraction
        refraction.ray.orig = incidence.ray.orig;
        refraction.ray.dir = rmagine::Vector::Zeros();
        refraction.velocity = v2;

        if(n1 > 0.0)
        {
            double n21 = n2 / n1;
            double angle_limit = 100.0;

            if(abs(n21) <= 1.0)
            {
                angle_limit = asin(n21);
            }

            if(incidence_angle <= angle_limit)
            {
                if(surface_normal.dot(incidence.ray.dir) > 0.0)
                {
                    surface_normal = -surface_normal;
                }
                if(n2 > 0.0)
                {
                    double n12 = n1 / n2;
                    double c = cos(incidence_angle);
                    refraction.ray.dir = incidence.ray.dir * n12 + surface_normal * (n12 * c - sqrt(1 - n12*n12 * ( 1 - c*c ) ) );
                }
            }
        }
        
        // energy
        double refraction_angle = acos((refraction.ray.dir).dot(-surface_normal));

        double rs = 0.0;
        double rp = 0.0;
        double eps = 0.0001;
        
        if(incidence_angle + refraction_angle < eps)
        {
            rs = (n1 - n2) / (n1 + n2);
            rp = rs;
        } else if(incidence_angle + refraction_angle > M_PI - eps) {
            rs = 1.0;
            rp = 1.0;
        } else {
            rs = - sin(incidence_angle - refraction_angle) / sin(incidence_angle + refraction_angle);
            rp = tan(incidence_angle - refraction_angle) / tan(incidence_angle + refraction_angle);
        }

        double Rs = rs * rs;
        double Rp = rp * rp;
        
        double Reff = incidence.polarization * Rs 
            + (1.0 - incidence.polarization) * Rp;
        
        double Teff = 1.0 - Reff;

        reflection.energy = Reff * incidence.energy;
        refraction.energy = Teff * incidence.energy;
    }

    // 4. returning signals
    if(reflection.energy > wave_energy_threshold)
    {
        // there is some energy reflected, so let it return
        if(reflection.material_id == material_id_air)
        {
            // 1. signal travelling back along the pass
            auto material = materials[refraction.material_id];
            double incidence_angle = get_incidence_angle(
                surface_normal, incidence);
             // 1. signal traveling over path
            double return_energy_path = back_reflection_shader(
                incidence_angle,
                reflection.energy,
                material.ambient, // ambient
                material.diffuse, // diffuse
                material.specular // specular
            );

            float time_back = incidence.time * 2.0;

            Signal sig;
            sig.time = incidence.time * 2.0;
            sig.strength = return_energy_path;
            signals[tid] = sig;
        }


        { // move with skip_dist and add to new waves
            reflection.ray.orig = reflection.ray.orig + reflection.ray.dir * skip_dist;
            reflection.time += skip_dist / reflection.velocity;
        }

        waves_new[reflectionid] = reflection;
        waves_new_mask[reflectionid] = 1;
    } else {
        waves_new_mask[reflectionid] = 0;
    }

    if(refraction.energy > wave_energy_threshold)
    {
        { // move with skip_dist and add to new waves
            refraction.ray.orig = refraction.ray.orig + refraction.ray.dir * skip_dist;
            refraction.time += skip_dist / refraction.velocity;
        }

        waves_new[refractionid] = refraction;
        waves_new_mask[refractionid] = 1;
    } else {
        waves_new_mask[refractionid] = 0;
    }


}

void propagate_waves(
    const rm::MemView<RadarMaterial, rm::VRAM_CUDA>& materials,
    const rm::MemView<int, rm::VRAM_CUDA>& object_materials,
    int material_id_air,

    const rm::MemView<DirectedWave, rm::VRAM_CUDA>& waves,
    const rm::MemView<uint8_t, rm::VRAM_CUDA>& hits,
    const rm::MemView<float, rm::VRAM_CUDA>& ranges,
    const rm::MemView<rm::Vector, rm::VRAM_CUDA>& normals,
    const rm::MemView<unsigned int, rm::VRAM_CUDA>& object_ids,
    
    rm::MemView<Signal, rm::VRAM_CUDA>& signals,
    rm::MemView<DirectedWave, rm::VRAM_CUDA>& waves_new,
    rm::MemView<uint8_t, rm::VRAM_CUDA>& waves_new_mask)
{
    propagate_waves_kernel<<<waves.size(),1>>>(
        materials.raw(),
        object_materials.raw(),
        material_id_air,

        waves.raw(), 
        waves.size(), 
        hits.raw(),
        ranges.raw(),
        normals.raw(),
        object_ids.raw(),
        
        signals.raw(),
        waves_new.raw(),
        waves_new_mask.raw()
    );
}


__global__ 
void move_waves_kernel(
    rm::Vector* origs,
    rm::Vector* dirs,
    DirectedWaveAttributes* attr,
    unsigned int n_waves,
    const float* ranges,
    const uint8_t* mask)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n_waves && mask[tid])
    {
        origs[tid] = origs[tid] + dirs[tid] * ranges[tid];
        attr[tid].time += ranges[tid] / attr[tid].velocity;
    }
}

void move_waves(
    rm::MemoryView<rm::Vector, rm::VRAM_CUDA>& wave_origs,
    rm::MemoryView<rm::Vector, rm::VRAM_CUDA>& wave_dirs,
    rm::MemoryView<DirectedWaveAttributes, rm::VRAM_CUDA>& wave_attributes,
    const rm::MemoryView<float, rm::VRAM_CUDA>& distances,
    const rm::MemoryView<uint8_t, rm::VRAM_CUDA>& mask)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (wave_origs.size() + blockSize - 1) / blockSize;

    move_waves_kernel<<<gridSize, blockSize>>>(
        wave_origs.raw(),
        wave_dirs.raw(),
        wave_attributes.raw(),
        wave_origs.size(),
        distances.raw(),
        mask.raw()
    );
}


__global__ 
void signal_shader_kernel(
    const RadarMaterial* materials,
    const int* object_materials,
    int material_id_air,

    const rm::Vector* dirs,
    const DirectedWaveAttributes* attr,
    unsigned int n_waves,

    const uint8_t* hits,
    const rm::Vector* surface_normals,
    const unsigned int* object_ids,

    Signal* signals
    )
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < n_waves && hits[tid])
    {
        const rm::Vector incidence_dir = dirs[tid];
        const DirectedWaveAttributes incidence_attr = attr[tid];
        rm::Vector surface_normal = surface_normals[tid];
        const unsigned int obj_id = object_ids[tid];

        // 2. split to reflection and refraction
        DirectedWaveAttributes reflection_attr = incidence_attr;
        DirectedWaveAttributes refraction_attr = incidence_attr;
        

        // if wave was in air, switch to new material
        // else if wave was in material, switch to air (is this right ?)
        if(incidence_attr.material_id == material_id_air)
        {
            refraction_attr.material_id = object_materials[obj_id];
        } else {
            refraction_attr.material_id = material_id_air;
        }

        float v_refraction = 1.0;

        if(incidence_attr.material_id != refraction_attr.material_id)
        {
            v_refraction = materials[refraction_attr.material_id].velocity;
        } else {
            v_refraction = incidence_attr.velocity;
        }

        // 3. fresnel
        {
            const double v1 = incidence_attr.velocity;
            const double v2 = v_refraction;

            const double n1 = v2;
            const double n2 = v1;

            double incidence_angle = acos((-incidence_dir).dot(surface_normal));
            
            // refraction
            rm::Vector refraction_dir = rmagine::Vector::Zeros();
            refraction_attr.velocity = v2;

            if(n1 > 0.0)
            {
                double n21 = n2 / n1;
                double angle_limit = 100.0;

                if(abs(n21) <= 1.0)
                {
                    angle_limit = asin(n21);
                }

                if(incidence_angle <= angle_limit)
                {
                    if(surface_normal.dot(incidence_dir) > 0.0)
                    {
                        surface_normal = -surface_normal;
                    }
                    if(n2 > 0.0)
                    {
                        double n12 = n1 / n2;
                        double c = cos(incidence_angle);
                        refraction_dir = incidence_dir * n12 
                                        + surface_normal * (n12 * c - sqrt(1 - n12*n12 * ( 1 - c*c ) ) );
                    }
                }
            }
            
            // // energy
            double refraction_angle = acos((refraction_dir).dot(-surface_normal));

            double rs = 0.0;
            double rp = 0.0;
            double eps = 0.0001;
            
            if(incidence_angle + refraction_angle < eps)
            {
                rs = (n1 - n2) / (n1 + n2);
                rp = rs;
            } else if(incidence_angle + refraction_angle > M_PI - eps) {
                rs = 1.0;
                rp = 1.0;
            } else {
                rs = -sin(incidence_angle - refraction_angle) / sin(incidence_angle + refraction_angle);
                rp = tan(incidence_angle - refraction_angle) / tan(incidence_angle + refraction_angle);
            }

            double Rs = rs * rs;
            double Rp = rp * rp;
            
            double Reff = incidence_attr.polarization * Rs 
                + (1.0 - incidence_attr.polarization) * Rp;

            reflection_attr.energy = Reff * incidence_attr.energy;
        }

        // there is some energy reflected, so let it return
        {
            // 1. signal travelling back along the pass
            auto material = materials[refraction_attr.material_id];
            double incidence_angle = get_incidence_angle(
                surface_normal, incidence_dir);
             // 1. signal traveling over path
            double return_energy_path = back_reflection_shader(
                incidence_angle,
                reflection_attr.energy,
                material.ambient, // ambient
                material.diffuse, // diffuse
                material.specular // specular
            );

            float time_back = incidence_attr.time * 2.0;

            Signal sig;
            sig.time = incidence_attr.time * 2.0;
            sig.strength = return_energy_path;
            signals[tid] = sig;
        }
    

    }
}






void signal_shader(
    const rm::MemView<RadarMaterial, rm::VRAM_CUDA>& materials,
    const rm::MemView<int, rm::VRAM_CUDA>& object_materials,
    int material_id_air,

    const rm::MemView<rm::Vector, rm::VRAM_CUDA>& dirs,
    const rm::MemView<DirectedWaveAttributes, rm::VRAM_CUDA>& attr,
    const rm::MemView<uint8_t, rm::VRAM_CUDA>& hits,
    const rm::MemView<rm::Vector, rm::VRAM_CUDA>& surface_normals,
    const rm::MemView<unsigned int, rm::VRAM_CUDA>& object_ids,

    rm::MemView<Signal, rm::VRAM_CUDA>& signals)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (dirs.size() + blockSize - 1) / blockSize;

    signal_shader_kernel<<<gridSize, blockSize>>>(
        materials.raw(),
        object_materials.raw(),
        material_id_air,

        dirs.raw(),
        attr.raw(),
        dirs.size(),
        
        hits.raw(),
        surface_normals.raw(),
        object_ids.raw(),

        signals.raw()
    );
}

__global__ 
void fresnel_split_kernel(
    const RadarMaterial* materials,
    const int* object_materials,
    int material_id_air,
    // INCIDENCE
    const rm::Vector* incidence_origs,
    const rm::Vector* incidence_dirs,
    const DirectedWaveAttributes* incidence_attrs,
    unsigned int n_incidences,
    const uint8_t* hits,
    const rm::Vector* surface_normals,
    const unsigned int* object_ids,
    // SPLIT
    rm::Vector* reflection_origs,
    rm::Vector* reflection_dirs,
    DirectedWaveAttributes* reflection_attrs,
    rm::Vector* refraction_origs,
    rm::Vector* refraction_dirs,
    DirectedWaveAttributes* refraction_attrs
    )
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n_incidences)
    {
        if(hits[tid] == 0)
        {
            reflection_origs[tid] = incidence_origs[tid];
            reflection_dirs[tid] = rm::Vector::Zeros();
            refraction_origs[tid] = incidence_origs[tid];
            refraction_dirs[tid] = rm::Vector::Zeros();
            return;
        }

        const rm::Vector incidence_orig = incidence_origs[tid];
        const rm::Vector incidence_dir = incidence_dirs[tid];
        const DirectedWaveAttributes incidence_attr = incidence_attrs[tid];
        rm::Vector surface_normal = surface_normals[tid];
        const unsigned int obj_id = object_ids[tid];

        // 2. split to reflection and refraction
        
        rm::Vector reflection_orig = incidence_orig;
        rm::Vector reflection_dir = rm::Vector::Zeros();
        DirectedWaveAttributes reflection_attr = incidence_attr;
        rm::Vector refraction_orig = incidence_orig;
        rm::Vector refraction_dir = rm::Vector::Zeros();
        DirectedWaveAttributes refraction_attr = incidence_attr;
        

        // if wave was in air, switch to new material
        // else if wave was in material, switch to air (is this right ?)
        if(incidence_attr.material_id == material_id_air)
        {
            refraction_attr.material_id = object_materials[obj_id];
        } else {
            refraction_attr.material_id = material_id_air;
        }

        float v_refraction = 1.0;

        if(incidence_attr.material_id != refraction_attr.material_id)
        {
            v_refraction = materials[refraction_attr.material_id].velocity;
        } else {
            v_refraction = incidence_attr.velocity;
        }

        // 3. fresnel
        {
            const double v1 = incidence_attr.velocity;
            const double v2 = v_refraction;

            const double n1 = v2;
            const double n2 = v1;

            double incidence_angle = acos((-incidence_dir).dot(surface_normal));

            // reflection
            reflection_dir = incidence_dir + surface_normal * 2.0 * (-surface_normal).dot(incidence_dir);

            // refraction
            refraction_attr.velocity = v2;

            if(n1 > 0.0)
            {
                double n21 = n2 / n1;
                double angle_limit = 100.0;

                if(abs(n21) <= 1.0)
                {
                    angle_limit = asin(n21);
                }

                if(incidence_angle <= angle_limit)
                {
                    if(surface_normal.dot(incidence_dir) > 0.0)
                    {
                        surface_normal = -surface_normal;
                    }
                    if(n2 > 0.0)
                    {
                        double n12 = n1 / n2;
                        double c = cos(incidence_angle);
                        refraction_dir = incidence_dir * n12 
                                        + surface_normal * (n12 * c - sqrt(1 - n12*n12 * ( 1 - c*c ) ) );
                    }
                }
            }
            
            // // energy
            double refraction_angle = acos((refraction_dir).dot(-surface_normal));

            double rs = 0.0;
            double rp = 0.0;
            double eps = 0.0001;
            
            if(incidence_angle + refraction_angle < eps)
            {
                rs = (n1 - n2) / (n1 + n2);
                rp = rs;
            } else if(incidence_angle + refraction_angle > M_PI - eps) {
                rs = 1.0;
                rp = 1.0;
            } else {
                rs = -sin(incidence_angle - refraction_angle) / sin(incidence_angle + refraction_angle);
                rp = tan(incidence_angle - refraction_angle) / tan(incidence_angle + refraction_angle);
            }

            double Rs = rs * rs;
            double Rp = rp * rp;
            
            double Reff = incidence_attr.polarization * Rs 
                + (1.0 - incidence_attr.polarization) * Rp;
            double Teff = 1.0 - Reff;

            reflection_attr.energy = Reff * incidence_attr.energy;
            refraction_attr.energy = Teff * incidence_attr.energy;

        }

    
        // move
        const float skip_dist = 0.001;

        { // move reflection ray a bit
            reflection_orig = reflection_orig + reflection_dir * skip_dist;
            reflection_attr.time += skip_dist / reflection_attr.velocity;
        }

        { // move refraction ray a bit
            refraction_orig = refraction_orig + refraction_dir * skip_dist;
            refraction_attr.time += skip_dist / refraction_attr.velocity;
        }

        // write back
        reflection_origs[tid] = reflection_orig;
        reflection_dirs[tid] = reflection_dir;
        reflection_attrs[tid] = reflection_attr;

        refraction_origs[tid] = refraction_orig;
        refraction_dirs[tid] = refraction_dir;
        refraction_attrs[tid] = refraction_attr;
    }
}


void fresnel_split(
    const rm::MemView<RadarMaterial, rm::VRAM_CUDA>& materials,
    const rm::MemView<int, rm::VRAM_CUDA>& object_materials,
    int material_id_air,
    // INCIDENCE
    const rm::MemView<rm::Vector, rm::VRAM_CUDA>& incidence_origs,
    const rm::MemView<rm::Vector, rm::VRAM_CUDA>& incidence_dirs,
    const rm::MemView<DirectedWaveAttributes, rm::VRAM_CUDA>& incidence_attrs,
    const rm::MemView<uint8_t, rm::VRAM_CUDA>& hits,
    const rm::MemView<rm::Vector, rm::VRAM_CUDA>& surface_normals,
    const rm::MemView<unsigned int, rm::VRAM_CUDA>& object_ids,
    // SPLIT
    rm::MemView<rm::Vector, rm::VRAM_CUDA>& reflection_origs,
    rm::MemView<rm::Vector, rm::VRAM_CUDA>& reflection_dirs,
    rm::MemView<DirectedWaveAttributes, rm::VRAM_CUDA>& reflection_attrs,
    rm::MemView<rm::Vector, rm::VRAM_CUDA>& refraction_origs,
    rm::MemView<rm::Vector, rm::VRAM_CUDA>& refraction_dirs,
    rm::MemView<DirectedWaveAttributes, rm::VRAM_CUDA>& refraction_attrs)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (incidence_origs.size() + blockSize - 1) / blockSize;

    fresnel_split_kernel<<<gridSize, blockSize>>>(
        materials.raw(),
        object_materials.raw(),
        material_id_air,
        // INCIDENCE
        incidence_origs.raw(),
        incidence_dirs.raw(),
        incidence_attrs.raw(),
        incidence_origs.size(),
        hits.raw(),
        surface_normals.raw(),
        object_ids.raw(),
        // SPLIT
        reflection_origs.raw(),
        reflection_dirs.raw(),
        reflection_attrs.raw(),
        refraction_origs.raw(),
        refraction_dirs.raw(),
        refraction_attrs.raw()
    );
}

__global__ 
void draw_signals_kernel(
    float* img,
    float* max_vals,
    unsigned int* signal_counts,
    unsigned int n_angles,
    unsigned int n_cells,
    const Signal* signals,
    const uint8_t* mask,
    unsigned int n_samples,
    const unsigned int denoising_type,
    const float* denoising_weights,
    unsigned int n_denoising_weights,
    int denoising_mode,
    float resolution)
{
    unsigned int angle_id = blockIdx.x * blockDim.x + threadIdx.x;
    // unsigned int n_signals = n_angles * n_samples;
    unsigned int img_offset = angle_id * n_cells;



    // angles are hid (horizontal)
    if(angle_id < n_angles)
    {
        float max_val = max_vals[angle_id];
        unsigned int signal_count = signal_counts[angle_id];

        // sample is vid (vertical)
        for(unsigned int sample_id=0; sample_id<n_samples; sample_id++)
        {
            const unsigned int signal_id = sample_id * n_angles + angle_id;
            if(mask[signal_id])
            {
                const Signal signal = signals[signal_id];

                const float half_time = signal.time / 2.0;
                const float signal_dist = 0.3 * half_time;

                const int cell = static_cast<int>(signal_dist / resolution);

                if(cell < n_cells)
                {
                    if(denoising_type > 0)
                    {
                        for(int vid = 0; vid < n_denoising_weights; vid++)
                        {
                            int glob_id = vid + cell - denoising_mode;
                            if(glob_id > 0 && glob_id < n_cells)
                            {
                                // TODO: check this
                                const float old_val = img[img_offset + glob_id];
                                const float new_val = old_val + signal.strength * denoising_weights[vid];
                                img[img_offset + glob_id] = new_val;

                                if(new_val > max_val)
                                {
                                    max_val = new_val;
                                }
                            }
                        }
                    } else {
                        // read 
                        // TODO: check this
                        const float old_val = img[img_offset + cell];
                        const float new_val = max(old_val, (float)signal.strength);
                        img[img_offset + cell] = new_val;

                        if(new_val > max_val)
                        {
                            max_val = new_val;
                        }
                    }

                    signal_count++;
                }

            }
        }

        max_vals[angle_id] = max_val;
        signal_counts[angle_id] = signal_count;
    }
}


void draw_signals(
    rm::MemView<float, rm::VRAM_CUDA>& img,
    rm::MemView<float, rm::VRAM_CUDA>& max_vals,
    rm::MemView<unsigned int, rm::VRAM_CUDA>& signal_counts,
    unsigned int n_angles,
    unsigned int n_cells,
    const rm::MemView<Signal, rm::VRAM_CUDA>& signals,
    const rm::MemView<uint8_t, rm::VRAM_CUDA>& mask,
    unsigned int n_samples,
    const unsigned int denoising_type,
    const rm::MemView<float, rm::VRAM_CUDA> denoising_weights,
    int denoising_mode,
    float resolution)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (n_angles + blockSize - 1) / blockSize;

    draw_signals_kernel<<<gridSize, blockSize>>>(
        img.raw(),
        max_vals.raw(),
        signal_counts.raw(),
        n_angles,
        n_cells,
        signals.raw(),
        mask.raw(),
        n_samples,
        denoising_type,
        denoising_weights.raw(),
        denoising_weights.size(),
        denoising_mode,
        resolution
    );

}

} // namespace radarays_ros

